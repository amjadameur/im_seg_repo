#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "seg_macros.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

typedef unsigned char uchar;
typedef unsigned int QRgb;

__global__ void seg_kernel(uchar *dev_gray_im, uchar *dev_green_im, int n_pixels) {

}

// Recupere la composante verte de l'image rgb, ceci permettra de ne pas
// sur-consommer la memoire du gpu
void get_green(uchar *green_im, QRgb *rgb_im, int width, int height) {

}

extern "C"
void gpu_im_seg(uchar *gray_im, QRgb *rgb_im, int width, int height) {
    uchar *green_im; // pointeur memoire cpu, pointera sur la composante verte de l'image rgb
    uchar *dev_gray_im, *dev_green_im; // pointeur memoire gpu
    int n_pixels = width*height;
    int size = n_pixels*sizeof(uchar);

    green_im = (uchar*) malloc(size);
    get_green(green_im, rgb_im, width, height);

    // Allocation memoire GPU
    hipMalloc((void**) &dev_green_im, size);
    hipMalloc((void**) &dev_gray_im, size);

    hipMemcpy(dev_green_im, green_im, size, hipMemcpyHostToDevice);

    // Recuperation des caractéristiques du gpu (nombre de threads par bloc)
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int blockSize = deviceProp.maxThreadsPerBlock;
    int numBlocks = (n_pixels - 1) / blockSize + 1;

    // Appel du kernel cuda pour realiser la segmentation
    seg_kernel<<<numBlocks, blockSize>>>(dev_gray_im, dev_green_im, n_pixels);
    hipDeviceSynchronize();

    // Recuperation de l'image segmentee
    hipMemcpy(gray_im, dev_gray_im, size, hipMemcpyDeviceToHost);

    // Desallocation memoire
    free(green_im);
    hipFree(dev_gray_im); hipFree(dev_green_im);
}

