#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "seg_macros.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

typedef unsigned char uchar;
typedef unsigned int QRgb;

__global__ void seg_kernel(uchar *dev_gray_im, uchar *dev_green_im, int n_pixels) {

}

// Recupere la composante verte de l'image rgb, ceci permettra de ne pas
// sur-consommer la memoire du gpu
void get_green(uchar *green_im, QRgb *rgb_im, int width, int height) {

}

extern "C"
void gpu_im_seg(uchar *gray_im, QRgb *rgb_im, int width, int height) {
}

